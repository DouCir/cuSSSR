/**
  *
  * Date         12/07/2009
  * ====
  *
  * Authors      Vincent Garcia
  * =======      Eric    Debreuve
  *              Michel  Barlaud
  *
  * Description  Given a reference point set and a query point set, the program returns
  * ===========  firts the distance between each query point and its k nearest neighbors in
  *              the reference point set, and second the indexes of these k nearest neighbors.
  *              The computation is performed using the API NVIDIA CUDA.
  *
  * Paper        Fast k nearest neighbor search using GPU
  * =====
  *
  * BibTeX       @INPROCEEDINGS{2008_garcia_cvgpu,
  * ======         author = {V. Garcia and E. Debreuve and M. Barlaud},
  *                title = {Fast k nearest neighbor search using GPU},
  *                booktitle = {CVPR Workshop on Computer Vision on GPU},
  *                year = {2008},
  *                address = {Anchorage, Alaska, USA},
  *                month = {June}
  *              }
  *
  */

#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//


/**
 * Given a matrix of size width*height, compute the square norm of each column.
 *
 * @param mat    : the matrix
 * @param width  : the number of columns for a colum major storage matrix
 * @param height : the number of rowm for a colum major storage matrix
 * @param norm   : the vector containing the norm of the matrix
 */
__global__ void cuComputeNorm(float *mat, int width, int pitch, int height, float *norm){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (xIndex<width){
        float val, sum=0;
        int i;
        for (i=0;i<height;i++){
            val  = mat[i*pitch+xIndex];
            sum += val*val;
        }
        norm[xIndex] = sum;
    }
}



/**
 * Given the distance matrix of size width*height, adds the column vector
 * of size 1*height to each column of the matrix.
 *
 * @param dist   : the matrix
 * @param width  : the number of columns for a colum major storage matrix
 * @param pitch  : the pitch in number of column
 * @param height : the number of rowm for a colum major storage matrix
 * @param vec    : the vector to be added
 */
__global__ void cuAddRNorm(float *dist, int width, int pitch, int height, float *vec){
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int xIndex = blockIdx.x * blockDim.x + tx;
    unsigned int yIndex = blockIdx.y * blockDim.y + ty;
    __shared__ float shared_vec[16];
    if (tx==0 && yIndex<height)
        shared_vec[ty]=vec[yIndex];
    __syncthreads();
    if (xIndex<width && yIndex<height)
        dist[yIndex*pitch+xIndex]+=shared_vec[ty];
}



/**
  * Gathers k-th smallest distances for each column of the distance matrix in the top.
  *
  * @param dist        distance matrix
  * @param dist_pitch  pitch of the distance matrix given in number of columns
  * @param ind         index matrix
  * @param ind_pitch   pitch of the index matrix given in number of columns
  * @param width       width of the distance matrix and of the index matrix
  * @param height      height of the distance matrix and of the index matrix
  * @param k           number of neighbors to consider
  */
__global__ void cuInsertionSort(float *dist, int dist_pitch, int *ind, int ind_pitch, int width, int height, int k){

	// Variables
    int l, i, j;
    float *p_dist;
	int   *p_ind;
    float curr_dist, max_dist;
    int   curr_row,  max_row;
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
    if (xIndex<width){
        
        // Pointer shift, initialization, and max value
        p_dist   = dist + xIndex;
		p_ind    = ind  + xIndex;
        max_dist = p_dist[0];
        p_ind[0] = 1;
        
        // Part 1 : sort kth firt elementZ
        for (l=1; l<k; l++){
            curr_row  = l * dist_pitch;
			curr_dist = p_dist[curr_row];
			if (curr_dist<max_dist){
                i=l-1;
				for (int a=0; a<l-1; a++){
					if (p_dist[a*dist_pitch]>curr_dist){
						i=a;
						break;
					}
				}
                for (j=l; j>i; j--){
					p_dist[j*dist_pitch] = p_dist[(j-1)*dist_pitch];
					p_ind[j*ind_pitch]   = p_ind[(j-1)*ind_pitch];
                }
				p_dist[i*dist_pitch] = curr_dist;
				p_ind[i*ind_pitch]   = l+1;
			}
			else
				p_ind[l*ind_pitch] = l+1;
			max_dist = p_dist[curr_row];
		}
        
        // Part 2 : insert element in the k-th first lines
        max_row = (k-1)*dist_pitch;
        for (l=k; l<height; l++){
			curr_dist = p_dist[l*dist_pitch];
			if (curr_dist<max_dist){
                i=k-1;
				for (int a=0; a<k-1; a++){
					if (p_dist[a*dist_pitch]>curr_dist){
						i=a;
						break;
					}
				}
                for (j=k-1; j>i; j--){
					p_dist[j*dist_pitch] = p_dist[(j-1)*dist_pitch];
					p_ind[j*ind_pitch]   = p_ind[(j-1)*ind_pitch];
                }
				p_dist[i*dist_pitch] = curr_dist;
				p_ind[i*ind_pitch]   = l+1;
                max_dist             = p_dist[max_row];
            }
        }
    }
}


/**
  * Computes the square root of the first line (width-th first element)
  * of the distance matrix.
  *
  * @param dist    distance matrix
  * @param width   width of the distance matrix
  * @param pitch   pitch of the distance matrix given in number of columns
  * @param k       number of neighbors to consider
  */
__global__ void cuAddQNormAndSqrt(float *dist, int width, int pitch, float *q, int k){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex<width && yIndex<k)
        dist[yIndex*pitch + xIndex] = sqrt(dist[yIndex*pitch + xIndex] + q[xIndex]);
}
